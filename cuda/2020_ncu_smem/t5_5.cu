/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
// compile with: nvcc -Xcompiler -fopenmp -o t5 t5.cu -O3 -lcublas -lineinfo

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <hipblas.h>

#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
            msg, hipGetErrorString(__err), \
            __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
  } while (0)

// cuBLAS API errors
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL

unsigned long long dtime_usec(unsigned long long start){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

// perform vector averaging over M vectors of length L,  followed by matrix-vector multiply
// repeat the above N times
// input vectors are stored as a set of N column-major matrices
// for each k in N: output[k] = matrix*input[k]
template <typename T>
void cpu_version1(T *input, T *output, T *matrix, int L, int M, int N){
#pragma omp parallel for
  for (int k = 0; k < N; k++){      // repeat the following, N times
    std::vector<T> v1(L);           // vector length of L
    for (int i = 0; i < M; i++)     // compute average vector over M input vectors
      for (int j = 0; j < L; j++)
        v1[j] += input[k*M*L+j*M+i];
    for (int j = 0; j < L; j++)
      v1[j] /= M;
    for (int i = 0; i < L; i++)     // matrix-vector multiply
      for (int j = 0; j < L; j++)
	output[i*N+k] += matrix[i*L+j]*v1[j];
  }
}

const int my_L = 1024; // maximum limit of 1024
const int my_M = 1024;
const int my_N = 1024;

template <typename T>
__global__ void gpu_version1(const T * __restrict__ input, T * __restrict__ output, const T * __restrict__ matrix, const int L, const int M, const int N){
  // parallelize over vector length
  __shared__ T smem[my_L];
  int idx = threadIdx.x;
  for (int k = 0; k < N; k++){
    T v1 = 0;
    for (int i = 0; i < M; i++)
      v1 += input[k*M*L+idx*M+i];
    v1 /= M;
    for (int i = 0; i < L; i++){
      smem[threadIdx.x] = v1 * matrix[i*L+idx];
      for (int s = blockDim.x>>1; s > 0; s>>=1){
        __syncthreads(); 
	if (threadIdx.x < s) smem[threadIdx.x] += smem[threadIdx.x+s];}
      if (!threadIdx.x) output[k+i*N] = smem[0];}
  }
}

template <typename T>
__global__ void gpu_version2(const T * __restrict__ input, T * __restrict__ output, const T * __restrict__ matrix, const int L, const int M, const int N){
  // parallelize threadIdx.x over vector length, and blockIdx.x across k (N)
  __shared__ T smem[my_L];
  int idx = threadIdx.x;
  int k = blockIdx.x;
    T v1 = 0;
    for (int i = 0; i < M; i++)
      v1 += input[k*M*L+idx*M+i];
    v1 /= M;
    for (int i = 0; i < L; i++){
      smem[threadIdx.x] = v1 * matrix[i*L+idx];
      for (int s = blockDim.x>>1; s > 0; s>>=1){
        __syncthreads(); 
	if (threadIdx.x < s) smem[threadIdx.x] += smem[threadIdx.x+s];}
      if (!threadIdx.x) output[k+i*N] = smem[0];}
}

template <typename T>
__global__ void gpu_version3(const T * __restrict__ input, T * __restrict__ output, const T * __restrict__ matrix, const int L, const int M, const int N){
  // parallelize threadIdx.x over vector length, and blockIdx.x across k (N)
  // do initial vector reduction via warp-stride loop
  __shared__ T smem[my_L];
  int idx = threadIdx.x;
  int idy = threadIdx.y;
  int id  = idy*warpSize+idx;
  int k = blockIdx.x;
  T v1;
  for (int y = threadIdx.y; y < L; y+=blockDim.y){ // vertical block-stride loop
    v1 = 0;
    for (int x = threadIdx.x; x < M; x+=warpSize)  // horizontal warp-stide loop
      v1 += input[k*M*L+y*M+x];
    for (int offset = warpSize>>1; offset > 0; offset >>= 1) // warp-shuffle reduction
       v1 += __shfl_down_sync(0xFFFFFFFF, v1, offset);
    if (!threadIdx.x) smem[y] = v1/M;}
  __syncthreads();
  v1 = smem[id];
  for (int i = 0; i < L; i++){                     // matrix-vector multiply
    __syncthreads();
    smem[id] = v1 * matrix[i*L+id];
    for (int s = (blockDim.x*blockDim.y)>>1; s > 0; s>>=1){
      __syncthreads();
      if (id < s) smem[id] += smem[id+s];}
    if (!id) output[k+i*N] = smem[0];}
}

template <typename T>
__global__ void gpu_version4(const T * __restrict__ input, T * __restrict__ output, const T * __restrict__ matrix, const int L, const int M, const int N){
  // parallelize threadIdx.x over vector length, and blockIdx.x across k (N)
  // do initial vector reduction via warp-stride loop
  __shared__ T smem[my_L];
  int idx = threadIdx.x;
  int idy = threadIdx.y;
  int id  = idy*warpSize+idx;
  int k = blockIdx.x;
  T v1;
  for (int y = threadIdx.y; y < L; y+=blockDim.y){ // vertical block-stride loop
    v1 = 0;
    for (int x = threadIdx.x; x < M; x+=warpSize)  // horizontal warp-stide loop
      v1 += input[k*M*L+y*M+x];
    for (int offset = warpSize>>1; offset > 0; offset >>= 1) // warp-shuffle reduction
       v1 += __shfl_down_sync(0xFFFFFFFF, v1, offset);
    if (!threadIdx.x) smem[y] = v1/M;}
  __syncthreads();
  v1 = smem[id];
  for (int i = 0; i < L; i++){                     // matrix-vector multiply
    T v2 = v1 * matrix[i*L+id];
// 1st warp-shuffle reduction
    for (int offset = warpSize>>1; offset > 0; offset >>= 1) 
       v2 += __shfl_down_sync(0xFFFFFFFF, v2, offset);
    if (idx == 0) smem[idy] = v2;
   __syncthreads(); // put warp results in shared mem
// hereafter, just warp 0
    if (idy == 0){
 // reload v2 from shared mem if warp existed
       v2 = (idx < ((blockDim.x*blockDim.y)>>5))?smem[idx]:0;
 // final warp-shuffle reduction
       for (int offset = warpSize>>1; offset > 0; offset >>= 1) 
          v2 += __shfl_down_sync(0xFFFFFFFF, v2, offset);}
    if (!id) output[k+i*N] = v2;}
}

template <typename T>
__global__ void gpu_version5(const T * __restrict__ input, T * __restrict__ output, const int L, const int M, const int N){
  // parallelize threadIdx.x over vector length, and blockIdx.x across k (N)
  // do initial vector reduction via warp-stride loop
  int k = blockIdx.x;
  T v1;
  for (int y = threadIdx.y; y < L; y+=blockDim.y){ // vertical block-stride loop
    v1 = 0;
    for (int x = threadIdx.x; x < M; x+=warpSize)  // horizontal warp-stide loop
      v1 += input[k*M*L+y*M+x];
    for (int offset = warpSize>>1; offset > 0; offset >>= 1) // warp-shuffle reduction
       v1 += __shfl_down_sync(0xFFFFFFFF, v1, offset);
    if (!threadIdx.x) output[k+y*N] = v1/M;}
}


typedef float ft;

int main(){
  ft *d_input, *h_input, *d_output, *h_outputc, *h_outputg, *d_matrix, *h_matrix, *d_result;
  int L = my_L; int M = my_M; int N = my_N;
  // host allocations
  h_input   = new ft[N*L*M];
  h_matrix  = new ft[L*L];
  h_outputg = new ft[N*L];
  h_outputc = new ft[N*L];
  // data initialization
  for (int i = 0; i < N*L*M; i++) h_input[i] = (rand()&1)+1;  // 1 or 2
  for (int i = 0; i < L*L; i++) h_matrix[i]  = (rand()&1)+1;  // 1 or 2
  // create result to test for correctness
  unsigned long long dt = dtime_usec(0);
  cpu_version1(h_input, h_outputc, h_matrix, L, M, N);
  dt = dtime_usec(dt);
  std::cout << "CPU execution time: " << dt/(float)USECPSEC << "s" << std::endl;
  // device allocations
  hipMalloc(&d_input, N*L*M*sizeof(ft));
  hipMalloc(&d_output,  N*L*sizeof(ft));
  hipMalloc(&d_matrix,  L*L*sizeof(ft));
  hipMalloc(&d_result,  N*L*sizeof(ft));
  cudaCheckErrors("hipMalloc failure");
  // copy input data from host to device
  hipMemcpy(d_input,  h_input,  N*L*M*sizeof(ft), hipMemcpyHostToDevice);
  hipMemcpy(d_matrix, h_matrix,   L*L*sizeof(ft), hipMemcpyHostToDevice);
  hipMemset(d_output, 0, N*L*sizeof(ft));
  cudaCheckErrors("cudaMemcpy/Memset failure");
  // cublas setup
  hipblasHandle_t h;
  ft alpha = 1.0;
  ft beta  = 0.0;
  hipblasStatus_t c_res = hipblasCreate(&h);
  if (c_res != HIPBLAS_STATUS_SUCCESS) {std::cout << "CUBLAS create error: " << _cudaGetErrorEnum(c_res) << std::endl; return 0;}
  // run on device and measure execution time
  dim3 block(32,32);
  dt = dtime_usec(0);
  gpu_version5<<<N, block>>>(d_input, d_output, L, M, N);
  cudaCheckErrors("kernel launch failure");
  c_res = hipblasSgemm(h, HIPBLAS_OP_T, HIPBLAS_OP_T,
                           N, N, L, &alpha,
                           d_matrix, L,
                           d_output, N, &beta,
                           d_result, N);
  if (c_res != HIPBLAS_STATUS_SUCCESS) {std::cout << "CUBLAS gemm error: " << _cudaGetErrorEnum(c_res) << std::endl; return 0;}
  hipDeviceSynchronize();
  cudaCheckErrors("execution failure");
  dt = dtime_usec(dt);
  hipMemcpy(h_outputg, d_result, N*L*sizeof(ft), hipMemcpyDeviceToHost);
  cudaCheckErrors("hipMemcpy failure");
  for (int i = 0; i < N; i++)
    for (int j = 0; j < L; j++) if (h_outputg[i+N*j] != h_outputc[j+N*i]) {std::cout << "Mismatch at " << i << " was: " << h_outputg[i] << " should be: " << h_outputc[i] << std::endl; return 0;}
  std::cout << "Kernel execution time: " << dt/(float)USECPSEC << "s" << std::endl;
  return 0;
}

