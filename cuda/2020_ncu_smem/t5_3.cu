/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
// compile with: nvcc -Xcompiler -fopenmp -o t5 t5.cu -O3 -lineinfo

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
            msg, hipGetErrorString(__err), \
            __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
  } while (0)


#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL

unsigned long long dtime_usec(unsigned long long start){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

// perform vector averaging over M vectors of length L,  followed by matrix-vector multiply
// repeat the above N times
// input vectors are stored as a set of N column-major matrices
// for each k in N: output[k] = matrix*input[k]
template <typename T>
void cpu_version1(T *input, T *output, T *matrix, int L, int M, int N){
#pragma omp parallel for
  for (int k = 0; k < N; k++){      // repeat the following, N times
    std::vector<T> v1(L);           // vector length of L
    for (int i = 0; i < M; i++)     // compute average vector over M input vectors
      for (int j = 0; j < L; j++)
        v1[j] += input[k*M*L+j*M+i];
    for (int j = 0; j < L; j++)
      v1[j] /= M;
    for (int i = 0; i < L; i++)     // matrix-vector multiply
      for (int j = 0; j < L; j++)
	output[i*N+k] += matrix[i*L+j]*v1[j];
  }
}

const int my_L = 1024; // maximum limit of 1024
const int my_M = 1024;
const int my_N = 1024;

template <typename T>
__global__ void gpu_version1(const T * __restrict__ input, T * __restrict__ output, const T * __restrict__ matrix, const int L, const int M, const int N){
  // parallelize over vector length
  __shared__ T smem[my_L];
  int idx = threadIdx.x;
  for (int k = 0; k < N; k++){
    T v1 = 0;
    for (int i = 0; i < M; i++)
      v1 += input[k*M*L+idx*M+i];
    v1 /= M;
    for (int i = 0; i < L; i++){
      smem[threadIdx.x] = v1 * matrix[i*L+idx];
      for (int s = blockDim.x>>1; s > 0; s>>=1){
        __syncthreads(); 
	if (threadIdx.x < s) smem[threadIdx.x] += smem[threadIdx.x+s];}
      if (!threadIdx.x) output[k+i*N] = smem[0];}
  }
}

template <typename T>
__global__ void gpu_version2(const T * __restrict__ input, T * __restrict__ output, const T * __restrict__ matrix, const int L, const int M, const int N){
  // parallelize threadIdx.x over vector length, and blockIdx.x across k (N)
  __shared__ T smem[my_L];
  int idx = threadIdx.x;
  int k = blockIdx.x;
    T v1 = 0;
    for (int i = 0; i < M; i++)
      v1 += input[k*M*L+idx*M+i];
    v1 /= M;
    for (int i = 0; i < L; i++){
      smem[threadIdx.x] = v1 * matrix[i*L+idx];
      for (int s = blockDim.x>>1; s > 0; s>>=1){
        __syncthreads(); 
	if (threadIdx.x < s) smem[threadIdx.x] += smem[threadIdx.x+s];}
      if (!threadIdx.x) output[k+i*N] = smem[0];}
}

template <typename T>
__global__ void gpu_version3(const T * __restrict__ input, T * __restrict__ output, const T * __restrict__ matrix, const int L, const int M, const int N){
  // parallelize threadIdx.x over vector length, and blockIdx.x across k (N)
  // do initial vector reduction via warp-stride loop
  __shared__ T smem[my_L];
  int idx = threadIdx.x;
  int idy = threadIdx.y;
  int id  = idy*warpSize+idx;
  int k = blockIdx.x;
  T v1;
  for (int y = threadIdx.y; y < L; y+=blockDim.y){ // vertical block-stride loop
    v1 = 0;
    for (int x = threadIdx.x; x < M; x+=warpSize)  // horizontal warp-stide loop
      v1 += input[k*M*L+y*M+x];
    for (int offset = warpSize>>1; offset > 0; offset >>= 1) // warp-shuffle reduction
       v1 += __shfl_down_sync(0xFFFFFFFF, v1, offset);
    if (!threadIdx.x) smem[y] = v1/M;}
  __syncthreads();
  v1 = smem[id];
  for (int i = 0; i < L; i++){                     // matrix-vector multiply
    __syncthreads();
    smem[id] = v1 * matrix[i*L+id];
    for (int s = (blockDim.x*blockDim.y)>>1; s > 0; s>>=1){
      __syncthreads();
      if (id < s) smem[id] += smem[id+s];}
    if (!id) output[k+i*N] = smem[0];}
}


typedef float ft;

int main(){
  ft *d_input, *h_input, *d_output, *h_outputc, *h_outputg, *d_matrix, *h_matrix;
  int L = my_L; int M = my_M; int N = my_N;
  // host allocations
  h_input   = new ft[N*L*M];
  h_matrix  = new ft[L*L];
  h_outputg = new ft[N*L];
  h_outputc = new ft[N*L];
  // data initialization
  for (int i = 0; i < N*L*M; i++) h_input[i] = (rand()&1)+1;  // 1 or 2
  for (int i = 0; i < L*L; i++) h_matrix[i]  = (rand()&1)+1;  // 1 or 2
  // create result to test for correctness
  unsigned long long dt = dtime_usec(0);
  cpu_version1(h_input, h_outputc, h_matrix, L, M, N);
  dt = dtime_usec(dt);
  std::cout << "CPU execution time: " << dt/(float)USECPSEC << "s" << std::endl;
  // device allocations
  hipMalloc(&d_input, N*L*M*sizeof(ft));
  hipMalloc(&d_output,  N*L*sizeof(ft));
  hipMalloc(&d_matrix,  L*L*sizeof(ft));
  cudaCheckErrors("hipMalloc failure");
  // copy input data from host to device
  hipMemcpy(d_input,  h_input,  N*L*M*sizeof(ft), hipMemcpyHostToDevice);
  hipMemcpy(d_matrix, h_matrix,   L*L*sizeof(ft), hipMemcpyHostToDevice);
  hipMemset(d_output, 0, N*L*sizeof(ft));
  cudaCheckErrors("cudaMemcpy/Memset failure");
  // run on device and measure execution time
  dim3 block(32,32);
  dt = dtime_usec(0);
  gpu_version3<<<N, block>>>(d_input, d_output, d_matrix, L, M, N);
  cudaCheckErrors("kernel launch failure");
  hipDeviceSynchronize();
  cudaCheckErrors("kernel execution failure");
  dt = dtime_usec(dt);
  hipMemcpy(h_outputg, d_output, N*L*sizeof(ft), hipMemcpyDeviceToHost);
  cudaCheckErrors("hipMemcpy failure");
  for (int i = 0; i < N*L; i++) if (h_outputg[i] != h_outputc[i]) {std::cout << "Mismatch at " << i << " was: " << h_outputg[i] << " should be: " << h_outputc[i] << std::endl; return 0;}
  std::cout << "Kernel execution time: " << dt/(float)USECPSEC << "s" << std::endl;
  return 0;
}

