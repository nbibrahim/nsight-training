/*
 * Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
// compile with: nvcc -Xcompiler -fopenmp -o t5 t5.cu -O3 -lineinfo

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
            msg, hipGetErrorString(__err), \
            __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
  } while (0)


#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL

unsigned long long dtime_usec(unsigned long long start){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

// perform vector averaging over M vectors of length L,  followed by matrix-vector multiply
// repeat the above N times
// input vectors are stored as a set of N column-major matrices
// for each k in N: output[k] = matrix*input[k]
template <typename T>
void cpu_version1(T *input, T *output, T *matrix, int L, int M, int N){
#pragma omp parallel for
  for (int k = 0; k < N; k++){      // repeat the following, N times
    std::vector<T> v1(L);           // vector length of L
    for (int i = 0; i < M; i++)     // compute average vector over M input vectors
      for (int j = 0; j < L; j++)
        v1[j] += input[k*M*L+j*M+i];
    for (int j = 0; j < L; j++)
      v1[j] /= M;
    for (int i = 0; i < L; i++)     // matrix-vector multiply
      for (int j = 0; j < L; j++)
	output[i*N+k] += matrix[i*L+j]*v1[j];
  }
}

const int my_L = 1024; // maximum 1024
const int my_M = 1024;
const int my_N = 1024;

template <typename T>
__global__ void gpu_version1(const T * __restrict__ input, T * __restrict__ output, const T * __restrict__ matrix, const int L, const int M, const int N){

  __shared__ T smem[my_L];
  size_t idx = ((size_t)blockIdx.x)*blockDim.x + threadIdx.x;
  for (int k = 0; k < N; k++){  // iterate over N data sets
    T v1 = 0;
    for (int i = 0; i < M; i++) // perform vector averaging
      v1 += input[k*M*L+idx*M+i];
    v1 /= M;
    for (int i = 0; i < L; i++){ // perform matrix-vector multiply
      __syncthreads();
      smem[threadIdx.x] = v1 * matrix[i*L+idx];
      for (int s = blockDim.x>>1; s > 0; s>>=1){
        __syncthreads(); 
	if (threadIdx.x < s) smem[threadIdx.x] += smem[threadIdx.x+s];}
      if (!threadIdx.x) output[k+i*N] = smem[0];}
  }
}

typedef float ft;

int main(){
  ft *d_input, *h_input, *d_output, *h_outputc, *h_outputg, *d_matrix, *h_matrix;
  int L = my_L; int M = my_M; int N = my_N;
  // host allocations
  h_input   = new ft[N*L*M];
  h_matrix  = new ft[L*L];
  h_outputg = new ft[N*L];
  h_outputc = new ft[N*L];
  // data initialization
  for (int i = 0; i < N*L*M; i++) h_input[i] = (rand()&1)+1;  // 1 or 2
  for (int i = 0; i < L*L; i++) h_matrix[i]  = (rand()&1)+1;  // 1 or 2
  // create result to test for correctness
  unsigned long long dt = dtime_usec(0);
  cpu_version1(h_input, h_outputc, h_matrix, L, M, N);
  dt = dtime_usec(dt);
  std::cout << "CPU execution time: " << dt/(float)USECPSEC << "s" << std::endl;
  // device allocations
  hipMalloc(&d_input, N*L*M*sizeof(ft));
  hipMalloc(&d_output,  N*L*sizeof(ft));
  hipMalloc(&d_matrix,  L*L*sizeof(ft));
  cudaCheckErrors("hipMalloc failure");
  // copy input data from host to device
  hipMemcpy(d_input,  h_input,  N*L*M*sizeof(ft), hipMemcpyHostToDevice);
  hipMemcpy(d_matrix, h_matrix,   L*L*sizeof(ft), hipMemcpyHostToDevice);
  hipMemset(d_output, 0, N*L*sizeof(ft));
  cudaCheckErrors("cudaMemcpy/Memset failure");
  // run on device and measure execution time
  dt = dtime_usec(0);
  gpu_version1<<<1, L>>>(d_input, d_output, d_matrix, L, M, N);
  cudaCheckErrors("kernel launch failure");
  hipDeviceSynchronize();
  cudaCheckErrors("kernel execution failure");
  dt = dtime_usec(dt);
  hipMemcpy(h_outputg, d_output, N*L*sizeof(ft), hipMemcpyDeviceToHost);
  cudaCheckErrors("hipMemcpy failure");
  for (int i = 0; i < N*L; i++) if (h_outputg[i] != h_outputc[i]) {std::cout << "Mismatch at " << i << " was: " << h_outputg[i] << " should be: " << h_outputc[i] << std::endl; return 0;}
  std::cout << "Kernel execution time: " << dt/(float)USECPSEC << "s" << std::endl;
  return 0;
}

